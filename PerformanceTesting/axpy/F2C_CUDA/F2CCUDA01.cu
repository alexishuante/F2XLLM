#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


__global__ void saxpy_parallel(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;

    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    saxpy_parallel<<<gridSize, blockSize>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}


extern void saxpy(int n, float a, float *x, float *y);

int main() {
    int n = 1000000; // Size of the arrays
    float a = 2.0f; // Scalar value for saxpy
    float *x, *y;

    // Allocate memory for x and y on the host
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize x and y arrays
    for(int i = 0; i < n; i++) {
        x[i] = 1.0f; // Example value
        y[i] = 2.0f; // Example value
    }

    // Warmup run
    saxpy(n, a, x, y);

    // Timing
    clock_t start, end;
    double total_time = 0.0;

    for(int i = 0; i < 10; i++) {
        start = clock();
        saxpy(n, a, x, y);
        end = clock();
        total_time += (double)(end - start) / CLOCKS_PER_SEC;
    }

    double average_time = total_time / 10;
    printf("Average Time: %f seconds\n", average_time);

    // Free allocated memory
    free(x);
    free(y);

    return 0;
}